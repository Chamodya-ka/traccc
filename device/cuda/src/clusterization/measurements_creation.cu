#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "measurements_creation.hpp"

namespace traccc::cuda {

__global__ 
void measurement_creation(cluster_container_types::const_view clusters_view,
                          measurement_container_view measurements_view)
{
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize device vectors
    const cluster_container_types::const_device clusters_device(
        clusters_view);
    device_measurement_container measurement_device(
        measurements_view);

    // Ignore if idx is out of range
    if (idx >= clusters_device.size())
        return;

    // items: cluster of cells at current idx
    // header: cluster_id object with the information about the cell
    // module
    const auto& cluster = clusters_device.get_items().at(idx);
    const cluster_id& cl_id = clusters_device.get_headers().at(idx);

    const vector2 pitch = detail::get_pitch(cl_id);
    const auto module_idx = cl_id.module_idx;

    scalar totalWeight = 0.;

    // To calculate the mean and variance with high numerical
    // stability we use a weighted variant of Welford's algorithm.
    // This is a single-pass online algorithm that works well for
    // large numbers of samples, as well as samples with very high
    // values.
    //
    // To learn more about this algorithm please refer to:
    // [1] https://doi.org/10.1080/00401706.1962.10490022
    // [2] The Art of Computer Programming, Donald E. Knuth, second
    //     edition, chapter 4.2.2.
    point2 mean = {0., 0.}, var = {0., 0.};

    // Should not happen
    assert(cluster.empty() == false);

    detail::calc_cluster_properties(cluster, cl_id, mean, var,
                                    totalWeight);

    if (totalWeight > 0.) {
        measurement m;
        // normalize the cell position
        m.local = mean;
        // normalize the variance
        m.variance[0] = var[0] / totalWeight;
        m.variance[1] = var[1] / totalWeight;
        // plus pitch^2 / 12
        m.variance = m.variance + point2{pitch[0] * pitch[0] / 12,
                                            pitch[1] * pitch[1] / 12};
        // @todo add variance estimation
        measurement_device.get_items().at(module_idx).push_back(m);
    }
}

void measurement_creation(measurement_container_view measurements_view,
                          cluster_container_types::const_view clusters_view)
{

    // The kernel execution range
    auto n_clusters = clusters_view.headers.size();

    // Calculate the execution NDrange for the kernel
    auto workGroupSize = 64;
    auto num = (n_clusters + workGroupSize - 1) / workGroupSize;
    dim3 nMeasurementCreationThreads = {64,1,1};
    dim3 nMeasurementCreationBlocks = {num,1,1};
    // Run the kernel
    measurement_creation<<<nMeasurementCreationBlocks,nMeasurementCreationThreads>>>(
        clusters_view,measurements_view);


}

}  // namespace traccc::cuda