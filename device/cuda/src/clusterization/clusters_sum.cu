#include "hip/hip_runtime.h"
#include "clusters_sum.hpp"
#include "traccc/cuda/utils/definitions.hpp"

namespace traccc::cuda {
namespace kernels {
__global__ void clusters_sum(
    cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    unsigned int &total_clusters,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view)
    {

        // Get the global index
        std::size_t  idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Initialize the data on the device
        cell_container_types::const_device cells_device(cells_view);

        // Ignore if idx is out of range
        if (idx < cells_device.size()) {

            // Get the cells from the current module
            const auto& cells = cells_device.at(idx).items;

            // Number of clusters that sparce_ccl will find for this
            // module
            unsigned int n_clusters = 0;

            // Vectors used for cluster indices found by sparse CCL
            vecmem::jagged_device_vector<unsigned int>
                device_sparse_ccl_indices(sparse_ccl_indices_view);
            auto cluster_indices =
                device_sparse_ccl_indices.at(idx);
            // Run the sparse_ccl algorithm
            printf("h1");
            detail::sparse_ccl(cells, cluster_indices, n_clusters);
            printf("h2");

            // Save the number of clusters found in this module at
            // the last, extra place in the indices vectors
            cluster_indices.back() = n_clusters;
            
            auto prefix_sum =
                vecmem::device_atomic_ref<unsigned int>(
                    total_clusters)
                    .fetch_add(n_clusters);

            // Save the current prefix sum at a correct index in a
            // vector
            vecmem::device_vector<std::size_t>
                device_cluster_prefix_sum(cluster_prefix_sum_view);
            device_cluster_prefix_sum[idx] = prefix_sum;

            // At last, fill also the "number of clusters per
            // module" for measurement creation buffer
            vecmem::device_vector<std::size_t>
                device_clusters_per_module(
                    clusters_per_module_view);
            device_clusters_per_module[idx] = n_clusters;
        }
            
  
    }
} //namespace kernels
void clusters_sum(
    const cell_container_types::host& cells_per_event,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    unsigned int total_clusters,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view,
    vecmem::memory_resource& resource) {


    // Execution size of the algorithm
    std::size_t n_modules = cells_per_event.size();
    // Calculate the execution NDrange for the kernel
    const unsigned int nClustersSumThreads = 64;
    const unsigned int nDClustersSumBlocks = (n_modules + nClustersSumThreads - 1) / nClustersSumThreads;

    
    // Get the view of the cells container
    auto cells_data = get_data(cells_per_event, &resource);
    cell_container_types::const_view cells_view(cells_data);
    // Launch clusters_sum kernel
    kernels::clusters_sum<<<nDClustersSumBlocks,nClustersSumThreads>>>
        (cells_view,sparse_ccl_indices_view,total_clusters,
        cluster_prefix_sum_view,clusters_per_module_view);  
    CUDA_ERROR_CHECK(hipPeekAtLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());     
    printf("5\n")   ;           
    }
    
} //namespace traccc::cuda