#include "hip/hip_runtime.h"

#include "clusters_sum.hpp"
#include "traccc/cuda/utils/definitions.hpp"
#include <assert.h>

#define CudaAssert( X ) if ( !(X) ) { printf( "Thread %d:%d failed assert at %s:%d!", blockIdx.x, threadIdx.x, __FILE__, __LINE__ ); return; }

namespace traccc::cuda {
namespace kernels {
__global__ void clusters_sum(
    cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    unsigned int& total_clusters,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view)
    {

        // Get the global index
        std::size_t  idx = threadIdx.x + blockIdx.x * blockDim.x;
        // Initialize the data on the device
        cell_container_types::const_device cells_device(cells_view);
        

        // Ignore if idx is out of range
        if (idx >= cells_device.size()) return;

        // Get the cells from the current module
        const auto& cells = cells_device.at(idx).items;

        // Vectors used for cluster indices found by sparse CCL
        vecmem::jagged_device_vector<unsigned int>
            device_sparse_ccl_indices(sparse_ccl_indices_view);
        auto cluster_indices =
            device_sparse_ccl_indices.at(idx);
        // Run the sparse_ccl algorithm
        unsigned int n_clusters = detail::sparse_ccl(cells, cluster_indices);
        // Save the number of clusters found in this module at
        // the last, extra place in the indices vectors
        cluster_indices.back() = n_clusters;
        
        auto prefix_sum =
            vecmem::device_atomic_ref<unsigned int>(
                total_clusters)
                .fetch_add(n_clusters);

        // Save the current prefix sum at a correct index in a
        // vector
        vecmem::device_vector<std::size_t>
            device_cluster_prefix_sum(cluster_prefix_sum_view);
        device_cluster_prefix_sum[idx] = prefix_sum;

        // At last, fill also the "number of clusters per
        // module" for measurement creation buffer
        vecmem::device_vector<std::size_t>
            device_clusters_per_module(
                clusters_per_module_view);
        device_clusters_per_module[idx] = n_clusters;
        
            
  
    }
} //namespace kernels
void clusters_sum(
    const cell_container_types::const_view& cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    unsigned int& total_clusters,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {

    auto n_modules = cells_view.headers.size();
    const unsigned int nClustersSumThreads = 64;
    const unsigned int nDClustersSumBlocks = (n_modules + nClustersSumThreads - 1) / nClustersSumThreads;

    // Launch clusters_sum kernel
    kernels::clusters_sum<<<nDClustersSumBlocks,nClustersSumThreads>>>
        (cells_view,sparse_ccl_indices_view,total_clusters,
        cluster_prefix_sum_view,clusters_per_module_view);    
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize()); 
    }
    
} //namespace traccc::cuda
