#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "cluster_counting.hpp"

namespace traccc::cuda {

__global__ void cluster_counting(std::size_t *n_modules_ptr,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<unsigned int> cluster_sizes_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view)
{   
    //auto id_x = item.get_global_id(0);
    std::size_t  id_x = threadIdx.x + blockIdx.x * blockDim.x; 
    //auto id_y = item.get_global_id(1);
    std::size_t  id_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Ignore if id_x is out of range
    if (id_x >= *n_modules_ptr)
        return;

    // Vectors used for cluster indices found by sparse CCL
    vecmem::jagged_device_vector<unsigned int>
        device_sparse_ccl_indices(sparse_ccl_indices_view);
    const auto& cluster_indices =
        device_sparse_ccl_indices.at(id_x);

    // Ignore if id_y is out of range (more than num of cells
    // for this module)
    if (id_y >= cluster_indices.size())
        return;

    // Number of clusters that sparce_ccl found for this module
    const unsigned int n_clusters = cluster_indices.back();

    // Get the prefix sum at this id_x to know where to write
    // clusters from this module to the cluster_container
    vecmem::device_vector<std::size_t>
        device_cluster_prefix_sum(cluster_prefix_sum_view);
    const std::size_t prefix_sum =
        device_cluster_prefix_sum[id_x];

    // Vector to fill in with the sizes of each cluster
    vecmem::device_vector<unsigned int> device_cluster_sizes(
        cluster_sizes_view);

    // Count the cluster sizes for each position
    unsigned int cindex = cluster_indices[id_y] - 1;
    if (cindex < n_clusters) {
        vecmem::device_atomic_ref<unsigned int>(
            device_cluster_sizes[prefix_sum + cindex])
            .fetch_add(1);
    }



}   

void cluster_counting(
    std::size_t num_modules,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<unsigned int> cluster_sizes_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    std::size_t cells_max, vecmem::memory_resource& resource)
    {

    // X dimension of the execution grid (the Y dim is the cells_max)
    auto n_modules = vecmem::make_unique_alloc<std::size_t>(resource);
    *n_modules = num_modules;


    //CUDA kernel dementions
    auto wGroupSize = 16;
    const unsigned int numGroupsX = (*n_modules + wGroupSize - 1) / wGroupSize;
    const unsigned int numGroupsY = (cells_max + wGroupSize - 1) / wGroupSize;
    dim3 nClusterCountThreads = {wGroupSize,wGroupSize,1}; // 
    dim3 nCLusterCountBlocks =  {numGroupsX * wGroupSize,numGroupsY * wGroupSize,1};
    
    auto n_modules_ptr = n_modules.get();
     
    cluster_counting<<<nCLusterCountBlocks,nClusterCountThreads>>>(
        n_modules_ptr,sparse_ccl_indices_view,cluster_sizes_view,
        cluster_prefix_sum_view);
    
}

}  // namespace traccc::cuda