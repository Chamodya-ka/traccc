#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "traccc/cuda/seeding/track_params_estimation.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// VecMem include(s).
#include <boost/interprocess/sync/named_mutex.hpp>
#include <vecmem/utils/cuda/copy.hpp>

namespace traccc {
namespace cuda {

/// Forward declaration of track parameter estimating kernel
/// The bound track parameters at the bottom spacepoints are obtained
///
/// @param seeds_view seeds found by seed finding
/// @param params_view vector of bound track parameters at the bottom
/// spacepoints
__global__ void track_params_estimating_kernel(
    spacepoint_container_types::const_view spacepoints_view,
    vecmem::data::vector_view<const seed> seeds_view,
    vecmem::data::vector_view<bound_track_parameters> params_view);

track_params_estimation::track_params_estimation(
    const traccc::memory_resource& mr)
    : m_mr(mr), logfile(NULL), mem(NULL) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

track_params_estimation::track_params_estimation(
    const traccc::memory_resource& mr, std::ofstream* logfile,
    unsigned char* mem)
    : m_mr(mr), logfile(logfile), mem(mem) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

host_bound_track_parameters_collection track_params_estimation::operator()(
    const spacepoint_container_types::const_view& spacepoints_view,
    const vecmem::data::vector_view<const seed>& seeds_view) const {

    // Get the size of the seeds view
    auto seeds_size = m_copy->get_size(seeds_view);

    return this->operator()(spacepoints_view, seeds_view, seeds_size);
}

host_bound_track_parameters_collection track_params_estimation::operator()(
    const spacepoint_container_types::buffer& spacepoints_buffer,
    const vecmem::data::vector_buffer<seed>& seeds_buffer) const {

    // Get the size of the seeds buffer
    auto seeds_size = m_copy->get_size(seeds_buffer);

    return this->operator()(spacepoints_buffer, seeds_buffer, seeds_size);
}

host_bound_track_parameters_collection track_params_estimation::operator()(
    const spacepoint_container_types::const_view& spacepoints_view,
    const vecmem::data::vector_view<const seed>& seeds_view,
    std::size_t seeds_size) const {

    /* struct mutex_remove
    {
        mutex_remove() {
    boost::interprocess::named_mutex::remove("tracK_param"); } ~mutex_remove(){
    boost::interprocess::named_mutex::remove("tracK_param"); } } remover; */
    boost::interprocess::named_mutex mutex_3(
        boost::interprocess::open_or_create, "tracK_param");

    // Create output host container
    host_bound_track_parameters_collection params(
        seeds_size, (m_mr.host ? m_mr.host : &(m_mr.main)));

    // Check if anything needs to be done.
    if (seeds_size == 0) {
        return params;
    }

    // Create device buffer for the parameters
    vecmem::data::vector_buffer<bound_track_parameters> params_buffer(
        seeds_size, m_mr.main);
    m_copy->setup(params_buffer);

    // -- Num threads
    // The dimension of block is the integer multiple of WARP_SIZE (=32)
    unsigned int num_threads = WARP_SIZE * 2;

    // -- Num blocks
    // The dimension of grid is number_of_seeds / num_threads + 1
    unsigned int num_blocks = seeds_size / num_threads + 1;

    mutex_3.lock();
    Sync::complete(mem);
    mutex_3.unlock();
    printf("Waiting track_params_estimating_kernel\n");
    Sync::wait_for_other_processes(mem);
    printf("Done\n");
    auto start_track_param_est = std::chrono::system_clock::now();
    // run the kernel
    track_params_estimating_kernel<<<num_blocks, num_threads>>>(
        spacepoints_view, seeds_view, params_buffer);
    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    auto end_track_param_est = std::chrono::system_clock::now();
    std::chrono::duration<double> time_end_track_param_est =
        end_track_param_est - start_track_param_est;
    *logfile << time_end_track_param_est.count() << ",";
    mutex_3.lock();
    Sync::reset_shared_mem(mem);
    printf("track_params_estimating_kernel Done\n");
    mutex_3.unlock();
    Sync::wait_for_reset(mem);
    printf("reset complete\n");
    // Copy the results back to the host
    (*m_copy)(params_buffer, params);

    return params;
}

__global__ void track_params_estimating_kernel(
    spacepoint_container_types::const_view spacepoints_view,
    vecmem::data::vector_view<const seed> seeds_view,
    vecmem::data::vector_view<bound_track_parameters> params_view) {

    // Get device container for input parameters
    const spacepoint_container_types::const_device spacepoints_device(
        spacepoints_view);
    vecmem::device_vector<const seed> seeds_device(seeds_view);
    device_bound_track_parameters_collection params_device(params_view);

    // vector index for threads
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // prevent overflow
    if (gid >= seeds_device.size()) {
        return;
    }

    // convenient assumption on bfield and mass
    // TODO: make use of bfield extension for the future
    vector3 bfield = {0, 0, 2};

    const auto& seed = seeds_device.at(gid);
    auto& param = params_device[gid].vector();

    // Get bound track parameter
    param =
        seed_to_bound_vector(spacepoints_device, seed, bfield, PION_MASS_MEV);
}

}  // namespace cuda
}  // namespace traccc
